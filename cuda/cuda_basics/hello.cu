
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ const char *STR = "HELLO WORLD!";
const char STR_LENGTH = 12;

__global__ void hello()
{
	printf("%d %c\n", blockIdx.x, STR[threadIdx.x % STR_LENGTH]);
}

int main(void)
{
	int num_threads = STR_LENGTH;
	int num_blocks = 2;
  // 2*12 = 24 threads
	hello<<<num_blocks,num_threads>>>();
	hipDeviceSynchronize();

	return 0;
}

