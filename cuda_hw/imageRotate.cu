#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <stdio.h>
#include "opencv2/opencv.hpp"
using namespace cv;
using namespace std;

__global__ void rotateKernel(uchar3 * const d_in, uchar3 * const d_out, 
								uint imgheight, uint imgwidth)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
    
    d_out[idx * imgheight + idy].x = d_in[idy * imgwidth + idx].x;
	d_out[idx * imgheight + idy].y = d_in[idy * imgwidth + idx].y;
	d_out[idx * imgheight + idy].z = d_in[idy * imgwidth + idx].z;
	//test 1 d_out[idx * imgheight + idy]
}
int main(void)
{
    //Mat srcImage = imread("input_images/e1.jpg");
    Mat srcImage = imread("input_images/lenna.png");

    const uint imgheight = srcImage.rows;
    const uint imgwidth = srcImage.cols;

	Mat rotateImage(imgwidth, imgheight, CV_8UC3);
    uchar3 *d_in;
    unsigned char *d_out;

    hipMalloc((void**)&d_in, imgheight*imgwidth*sizeof(uchar3));
    hipMemcpy(d_in, srcImage.data, imgheight*imgwidth*sizeof(uchar3), hipMemcpyHostToDevice);
    
    dim3 threadsPerBlock(32, 32);
    dim3 blocksPerGrid((imgwidth + threadsPerBlock.x - 1) / threadsPerBlock.x,
         (imgheight + threadsPerBlock.y - 1) / threadsPerBlock.y);

	uchar3 *d_out2;
	hipMalloc((void**)&d_out2, imgheight*imgwidth*sizeof(uchar3));
	rotateKernel<< <blocksPerGrid, threadsPerBlock>> >(d_in, d_out2, imgheight, imgwidth);
	hipMemcpy(rotateImage.data, d_out2, imgheight*imgwidth*sizeof(uchar3), hipMemcpyDeviceToHost);
	hipFree(d_in);
	hipFree(d_out2);
	imwrite("output_images/lennaRotateImage.jpg",rotateImage);
	
    return 0;

}
