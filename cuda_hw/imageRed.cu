#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <stdio.h>
#include "opencv2/opencv.hpp"
using namespace cv;
using namespace std;

#define RED_SIZE 3 
// 5:34 pm

__global__ void redKernel(uchar3 * const d_in, uchar3 * const d_out, 
								uint imgheight, uint imgwidth)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;  // column
    const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;  // row

	if (idx < imgwidth && idy < imgheight) {
		int pixVal1 = 0;
		int pixVal2 = 0;
		int pixVal3 = 0;
		int pixel_index = 0;
		int redPixel =0;

		// Get the average of the surrounding BLUR_SIZE x BLUR_SIZE box
		for (int redrow = -RED_SIZE; redrow < RED_SIZE + 1; ++redrow) {
			for (int redcol = -RED_SIZE; redcol < RED_SIZE + 1; ++redcol) {
				int currow = idy + redrow;
				int curcol = idx + redcol;

				// Verify we have a valid image pixel
				if (currow > -1 && currow < imgheight && curcol > -1 && curcol < imgwidth) {
					
					pixVal1 = d_in[currow * imgwidth + curcol].x;
					pixVal2 = d_in[currow * imgwidth + curcol].y;
					pixVal3 = d_in[currow * imgwidth + curcol].z;

					if(pixVal1 > redPixel){
						redPixel = pixVal1;
					}

					pixel_index++;
				}
			}
		}
		// Write our new pixel value out
		d_out[idy * imgwidth + idx].x = (unsigned char)(redPixel);
		d_out[idy * imgwidth + idx].y = (unsigned char)(pixVal2);
		d_out[idy * imgwidth + idx].z = (unsigned char)(pixVal3);
	}
}
int main(void)
{
    Mat srcImage = imread("input_images/lenna.png");
    const uint imgheight = srcImage.rows;
    const uint imgwidth = srcImage.cols;


    uchar3 *d_in;

    hipMalloc((void**)&d_in, imgheight*imgwidth*sizeof(uchar3));

    hipMemcpy(d_in, srcImage.data, imgheight*imgwidth*sizeof(uchar3), hipMemcpyHostToDevice);
    
    dim3 threadsPerBlock(32, 32);
    dim3 blocksPerGrid((imgwidth + threadsPerBlock.x - 1) / threadsPerBlock.x,
         (imgheight + threadsPerBlock.y - 1) / threadsPerBlock.y);
  
	Mat redImage(imgheight, imgwidth, CV_8UC3);
	uchar3 *d_out2;
	hipMalloc((void**)&d_out2, imgheight*imgwidth*sizeof(uchar3));
	redKernel<< <blocksPerGrid, threadsPerBlock>> >(d_in, d_out2, imgheight, imgwidth);
	hipMemcpy(redImage.data, d_out2, imgheight*imgwidth*sizeof(uchar3), hipMemcpyDeviceToHost);
	hipFree(d_in);
	hipFree(d_out2);
	imwrite("output_images/lennaRedImage.jpg",redImage);
	
    return 0;

}
