
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 70
__global__ void factorial(int *a, int *b, int *c)
{
   int index = blockIdx.x * blockDim.x + threadIdx.x;
   int number = a[index];
   int factorial = 1;
   // 5 = 5*4*3*2*1;
   while(number>1)
   {
      factorial = factorial * number;
      number = number - 1;
   }

   c[index] = factorial;

  /*
         c[0] = a[0]!
         c[1] = a[1]!

         c[63] = a[63]!
   */
}

__global__ void add(int *a, int *b, int *c)
{
    /* insert code to calculate the index properly using blockIdx.x, blockDim.x, threadIdx.x */
	int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index < N)
  	 c[index] = a[index] + b[index];
}

/* experiment with N */
/* how large can it be? */
//#define N (2048*2048)
//#define N 70
#define THREADS_PER_BLOCK 8

int main()
{
  int *a, *b, *c; // Arrays in CPU
	int *d_a, *d_b, *d_c;  // Arrays to be allocated in a GPU
	int size = N * sizeof( int );

	/* allocate space for device copies of a, b, c */

	hipMalloc( (void **) &d_a, size );
	hipMalloc( (void **) &d_b, size );
	hipMalloc( (void **) &d_c, size );

	/* allocate space for host copies of a, b, c and setup input values */

	a = (int *)malloc( size );
	b = (int *)malloc( size );
	c = (int *)malloc( size );

	for( int i = 0; i < N; i++ )
	{
		a[i] = b[i] = i;
		c[i] = 0;
	}

	/* copy inputs to device */
	/* fix the parameters needed to copy data to the device */
	hipMemcpy( d_a, a, size, hipMemcpyHostToDevice );
	hipMemcpy( d_b, b, size, hipMemcpyHostToDevice );

	/* launch the kernel on the GPU */
	/* insert the launch parameters to launch the kernel properly using blocks and threads */ 
	add<<< (N + (THREADS_PER_BLOCK-1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>( d_a, d_b, d_c );
  //  10/3 = 3 blocks are not enough= 3*3 = 9
  //  ceiling   = 4 blocks are created = (4*3) = 12 threads

	/* copy result back to host */
	/* fix the parameters needed to copy data back to the host */
	hipMemcpy( c, d_c, size, hipMemcpyDeviceToHost );

  int index = 0; 
	printf( "c[%d] = %d\n",index+0, c[0] );
	printf( "c[%d] = %d\n",index+1, c[1] );
	printf( "c[%d] = %d\n",index+2, c[2] );
	printf( "c[%d] = %d\n",index+3, c[3] );
	printf( "c[%d] = %d\n",index+4, c[4] );

	printf( "c[%d] = %d\n",N-1, c[N-1] );

	/* clean up */

	free(a);
	free(b);
	free(c);
	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );
	
	return 0;
} /* end main */
